#include "hip/hip_runtime.h"
#include "mandelbrot.h"

#include <hip/hip_runtime.h>

__global__ void mandelbrot_kernel(float h, float v, int* iter_buffer, int width, int height) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i >= width || j >= height) { return; }

	float x0 = (h * (float)i) - 2.00f;
	float y0 = (v * (float)j) - 1.12f;

	float x = 0.f;
	float y = 0.f;

	int iteration = 0;
	while (x*x + y*y <= 2.f*2.f && iteration < MANDELBROT_MAX_ITERATIONS) {
		float x_temp = x*x - y*y + x0;
		y = 2.f*x*y + y0;
		x = x_temp;
		iteration++;
	}

	iter_buffer[j*width + i] = iteration;
}

void mandelbrot_gpu(int* iter_buffer, int width, int height) {
	hipError_t err;

	size_t buffer_size = width * height * sizeof(int);
	int* iter_buffer_d;
	err = hipMalloc((void**)&iter_buffer_d, buffer_size);
	if (err != hipSuccess) { return; }

	float h = ((0.47f + 2.00f) / (float)(width));
	float v = ((1.12f + 1.12f) / (float)(height));

	dim3 block(32, 32);
	dim3 grid((width + block.x-1) / block.x, (height + block.y - 1) / block.y);
	mandelbrot_kernel<<<grid, block>>>(h, v, iter_buffer_d, width, height);

	err = hipGetLastError();
	if (err != hipSuccess) { goto error; }
	
	err = hipMemcpy(iter_buffer, iter_buffer_d, buffer_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) { goto error; }

error:
	// Does not really make sense to check the error here at the end if we do not propagate the error :)
	err = hipFree(iter_buffer_d);
	if (err != hipSuccess) { return; }
}